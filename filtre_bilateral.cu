
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <cstring>

__device__ float distance_spatiale(int x1, int y1, int x2, int y2) {
    return sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
}

__device__ float difference_intensite(unsigned char* image, int largeur, int hauteur, int x1, int y1, int x2, int y2) {
    return fabsf(image[y1 * largeur + x1] - image[y2 * largeur + x2]);
}

__global__ void appliquer_filtre_bilateral(unsigned char* image, unsigned char* image_filtrée, int largeur, int hauteur, float sigma_spatial, float sigma_intensite) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < largeur && y < hauteur) {
        float somme_poids = 0.0f;
        float somme_pixels = 0.0f;

        // Chaque thread calcule un poids pour un voisin
        int i = threadIdx.x - 3; // Les indices de la fenêtre sont de -3 à +3 (7x7 voisins)
        int j = threadIdx.y - 3;

        if (i >= -3 && i <= 3 && j >= -3 && j <= 3) {
            int x_voisin = x + i;
            int y_voisin = y + j;

            if (x_voisin >= 0 && x_voisin < largeur && y_voisin >= 0 && y_voisin < hauteur) {
                // Calcul du poids spatial et du poids d'intensité
                float poids_spatial = expf(-0.5f * (distance_spatiale(x, y, x_voisin, y_voisin) / sigma_spatial) * (distance_spatiale(x, y, x_voisin, y_voisin) / sigma_spatial));
                float poids_intensite = expf(-0.5f * (difference_intensite(image, largeur, hauteur, x, y, x_voisin, y_voisin) / sigma_intensite) * (difference_intensite(image, largeur, hauteur, x, y, x_voisin, y_voisin) / sigma_intensite));
                float poids = poids_spatial * poids_intensite;

                // Accumulation des poids et des pixels (en utilisant les variables locales dans chaque thread)
                somme_poids += poids; 
                somme_pixels += image[y_voisin * largeur + x_voisin] * poids; 
            }
        }

        // Si tous les voisins ont été traités, calcul du pixel filtré dans le thread central
        if (threadIdx.x == 3 && threadIdx.y == 3) {
            image_filtrée[y * largeur + x] = (unsigned char)(somme_pixels / somme_poids);
        }
    }
}

void ecrire_BMP(const char* chemin, unsigned char* image, int largeur, int hauteur) {
    FILE* fichier = fopen(chemin, "wb");
    if (!fichier) {
        std::cerr << "Erreur d'ouverture du fichier !" << std::endl;
        return;
    }

    unsigned char header[54] = {'B', 'M', 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, 0, 0, 0, 0, 1, 0, 8, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    
    unsigned int fileSize = 54 + 256 * 4 + largeur * hauteur;
    *(unsigned int*)&header[2] = fileSize;
    *(unsigned int*)&header[18] = largeur;
    *(unsigned int*)&header[22] = hauteur;
    *(unsigned int*)&header[34] = 256; // Nombre de couleurs
    *(unsigned int*)&header[38] = 2835; // Résolution X
    *(unsigned int*)&header[42] = 2835; // Résolution Y

    fwrite(header, 1, 54, fichier);

    for (int i = 0; i < largeur * hauteur; ++i) {
        unsigned char couleur[4] = {image[i], image[i], image[i], 0};
        fwrite(couleur, 1, 4, fichier);
    }

    fclose(fichier);
}

void charger_image(const char* chemin, unsigned char* image, int largeur, int hauteur) {
    FILE* fichier = fopen(chemin, "rb");
    if (!fichier) {
        std::cerr << "Erreur d'ouverture du fichier image !" << std::endl;
        return;
    }

    fseek(fichier, 54, SEEK_SET);  // Sauter l'en-tête BMP
    fread(image, sizeof(unsigned char), largeur * hauteur, fichier);
    fclose(fichier);
}

int main() {
    const int largeur = 512;
    const int hauteur = 512;
    unsigned char* image = new unsigned char[largeur * hauteur];
    unsigned char* image_filtrée = new unsigned char[largeur * hauteur];

    // Charger l'image "lena_gray.bmp" depuis le dossier "/workspace/TP_cuda/"
    charger_image("/workspace/TP_cuda/lena_gray.bmp", image, largeur, hauteur);

    // Appliquer le filtre bilatéral
    unsigned char* d_image;
    unsigned char* d_image_filtrée;
    hipMalloc(&d_image, largeur * hauteur * sizeof(unsigned char));
    hipMalloc(&d_image_filtrée, largeur * hauteur * sizeof(unsigned char));

    hipMemcpy(d_image, image, largeur * hauteur * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(7, 7); // Chaque thread dans la fenêtre de 7x7
    dim3 numBlocks((largeur + 6) / 7, (hauteur + 6) / 7); // Nombre de blocs pour couvrir l'image
    appliquer_filtre_bilateral<<<numBlocks, threadsPerBlock>>>(d_image, d_image_filtrée, largeur, hauteur, 1.0f, 50.0f);

    hipMemcpy(image_filtrée, d_image_filtrée, largeur * hauteur * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Sauvegarder l'image filtrée dans un fichier BMP
    ecrire_BMP("image_filtree.bmp", image_filtrée, largeur, hauteur);

    // Libérer la mémoire
    delete[] image;
    delete[] image_filtrée;
    hipFree(d_image);
    hipFree(d_image_filtrée);

    return 0;
}

